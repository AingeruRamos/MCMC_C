#include "hip/hip_runtime.h"
// Copyright Notice ===========================================================
//
// main.cu, Copyright (c) 2023 Aingeru Ramos
//
// All Rights Reserved ========================================================
//
// This file is part of MCMC_C software project.
//
// MCMC_C is propietary software. The author has all the rights to the work.
// No third party may make use of this work without explicit permission of the author.
//
// ============================================================================

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "../header/constants.h"
#include "../header/ising.h"
#include "../header/mcmc.h"

#define _CUDA(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//-----------------------------------------------------------------------------|
//-----------------------------------------------------------------------------|

__global__ void cuda_init_results(MODEL_RESULTS* device_results) {
    initialize_results<MODEL_RESULTS>(device_results);
}

__global__ void cuda_init_replicas(MODEL_NAME* device_replicas, int* device_rands, MODEL_RESULTS* device_results) {
    initialize_replicas<MODEL_NAME, MODEL_RESULTS>(device_replicas, device_rands, device_results);
}

__global__ void cuda_init_temps(double* device_temps) {
    initialize_temps(device_temps);
}

__global__ void cuda_init_swaps(int* device_n_swaps, Swap*** device_swap_planning) {
    initialize_swaps(device_n_swaps, device_swap_planning);
}

__global__ void cuda_run_iteration(MODEL_NAME* device_replicas, double* device_temps) {
    int replica_id = threadIdx.x + blockDim.x * threadIdx.y;
    MODEL_NAME* replica = &device_replicas[replica_id];
    double temp = device_temps[replica_id];

    MCMC_iteration<MODEL_NAME>(replica, temp);
}

__global__ void cuda_run_swaps(MODEL_NAME* device_replicas, double* device_temps, int* device_n_swaps, Swap*** device_swap_planning, int iteration) {
    int swap_index=threadIdx.x + blockDim.x * threadIdx.y;

    if(swap_index < device_n_swaps[iteration-1]) {
        Swap* swap = device_swap_planning[iteration-1][swap_index];
        double swap_prob = get_swap_prob<MODEL_NAME>(swap, device_replicas, device_temps);

        double r = device_replicas[swap->_swap_candidate_1]._rand_gen.rand_uniform();
        if(r < swap_prob) {
            double aux_temp = device_temps[swap->_swap_candidate_1];
            device_temps[swap->_swap_candidate_1] = device_temps[swap->_swap_candidate_2];
            device_temps[swap->_swap_candidate_2] = aux_temp;

            MODEL_RESULTS* aux_results = device_replicas[swap->_swap_candidate_1]._results;
            device_replicas[swap->_swap_candidate_1]._results = device_replicas[swap->_swap_candidate_2]._results;
            device_replicas[swap->_swap_candidate_2]._results = aux_results;
            swap->_accepted = true;
        }
    }
}

__global__ void cuda_print_swaps(int* device_n_swaps, Swap*** device_swap_planning) {
    print_swaps(device_n_swaps, device_swap_planning);
}

__global__ void cuda_print(MODEL_RESULTS* device_results, int replica_id) {
    print_result(&device_results[replica_id]);
}

//-----------------------------------------------------------------------------|
//-----------------------------------------------------------------------------|

void option_enabeler(int argc, char** argv);

int DEBUG_FLOW = 0;
int DEBUG_NO_SWAPS = 0;
int DEBUG_NO_RESULTS = 0;

int main(int argc, char** argv) {

    srand(time(NULL));

    option_enabeler(argc, argv);

//-----------------------------------------------------------------------------|
//-----------------------------------------------------------------------------|

    time_t begin_init = time(NULL);

    int NUM_BLOCKS = ((TOTAL_REPLICAS)/1024)+1; //* 1024 is the number of thread per block
    int NUM_THREADS = (TOTAL_REPLICAS)%1024;

    int* device_rands;
    _CUDA(hipMalloc((void**)&device_rands, TOTAL_REPLICAS*sizeof(int)));

    if(DEBUG_FLOW) { printf("Device -> Rands: OK\n"); }

    MODEL_RESULTS* device_results;
    _CUDA(hipMalloc((void**)&device_results, TOTAL_REPLICAS*sizeof(MODEL_RESULTS)));
    cuda_init_results<<<1,1>>>(device_results);

    if(DEBUG_FLOW) { printf("Device -> Results: OK\n"); }

    MODEL_NAME* device_replicas;
    _CUDA(hipMalloc((void**)&device_replicas, TOTAL_REPLICAS*sizeof(MODEL_NAME)));
    cuda_init_replicas<<<1,1>>>(device_replicas, device_rands, device_results);

    _CUDA(hipFree(device_rands));

    if(DEBUG_FLOW) { printf("Device -> Replicas: OK\n"); }

    double* device_temps;
    _CUDA(hipMalloc((void**)&device_temps, TOTAL_REPLICAS*sizeof(double)));
    cuda_init_temps<<<1,1>>>(device_temps);

    if(DEBUG_FLOW) { printf("Device -> Temps: OK\n"); }

    int* device_n_swaps;
    Swap*** device_swap_planning;

    if(SWAP_ACTIVE) {
        _CUDA(hipMalloc((void**)&device_n_swaps, N_ITERATIONS*sizeof(int)));
        _CUDA(hipMalloc((void**)&device_swap_planning, N_ITERATIONS*sizeof(Swap**)));

        cuda_init_swaps<<<1,1>>>(device_n_swaps, device_swap_planning);
    }

    if(DEBUG_FLOW) { printf("Device -> Swaps: OK\n"); }

    time_t end_init = time(NULL);

//-----------------------------------------------------------------------------|
//-----------------------------------------------------------------------------|

    time_t begin_exec = time(NULL);

    if(DEBUG_FLOW) { printf("Executing\n"); }

    for(int iteration=1; iteration<N_ITERATIONS; iteration++) {
        cuda_run_iteration<<<NUM_BLOCKS, NUM_THREADS>>>(device_replicas, device_temps);

        if(DEBUG_FLOW) { printf("Replicas: OK\n"); }

        if(SWAP_ACTIVE) {
            cuda_run_swaps<<<NUM_BLOCKS, NUM_THREADS>>>(device_replicas, device_temps, device_n_swaps, device_swap_planning, iteration);
        }
    }

    time_t end_exec = time(NULL);

//-----------------------------------------------------------------------------|
//-----------------------------------------------------------------------------|

    time_t begin_print = time(NULL);

    printf("%d,%d\n", NUM_BLOCKS, NUM_THREADS); // SIMULATION CONSTANTS
    printf("%d\n", N_ITERATIONS);
    printf("%d\n", SWAP_ACTIVE);
    printf("%f,%f,%f,%d\n", INIT_TEMP, END_TEMP, TEMP_STEP, TOTAL_REPLICAS);

    printf("#\n"); // MODEL CONSTANTS
    
    printf("%d\n", N_ROW);
    printf("%d\n", N_COL);
    printf("%f\n", SPIN_PLUS_PERCENTAGE);

    printf("#\n");

    if(!DEBUG_NO_SWAPS && SWAP_ACTIVE) { // SWAP PLANNING (ACCEPTED)
        cuda_print_swaps<<<1,1>>>(device_n_swaps, device_swap_planning);
        hipDeviceSynchronize();
    }

    printf("#\n"); // RESULTS

    if(!DEBUG_NO_RESULTS) {
        for(int replica_id=0; replica_id<TOTAL_REPLICAS; replica_id++) {
                cuda_print<<<1,1>>>(device_results, replica_id);
                hipDeviceSynchronize();
                printf("#\n");
        }
    }

    time_t end_print = time(NULL);

//-----------------------------------------------------------------------------|
//-----------------------------------------------------------------------------|

    double total_init = (double) (end_init-begin_init);
    double total_exec = (double) (end_exec-begin_exec);
    double total_print = (double) (end_print-begin_print);

    printf("%f\n", total_init); // TIME
    printf("%f\n", total_exec);
    printf("%f\n", total_print);

//-----------------------------------------------------------------------------|
//-----------------------------------------------------------------------------|

    _CUDA(hipFree(device_results));
    _CUDA(hipFree(device_replicas));
    _CUDA(hipFree(device_temps));

    if(SWAP_ACTIVE) {
        _CUDA(hipFree(device_n_swaps));
        _CUDA(hipFree(device_swap_planning)); //TODO Improve "swap_planning" memory free
    }

    return 0;
}

//-----------------------------------------------------------------------------|
//-----------------------------------------------------------------------------|

void option_enabeler(int argc, char** argv) {
    for(int i=1; i<argc; i++) {

        if(strcmp(argv[i], "-df") == 0) {
            DEBUG_FLOW = 1;
            continue;
        } else if(strcmp(argv[i], "-nr") == 0) {
            DEBUG_NO_RESULTS = 1;
            continue;
        } else if(strcmp(argv[i], "-ns") == 0) {
            DEBUG_NO_SWAPS = 1;
            continue;
        }
    }
}